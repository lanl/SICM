
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <ctype.h>
#include <unistd.h>
#include <time.h>
#include <math.h>
#include <sched.h>
#include <sys/time.h>
#include "numa.h"

__global__
void saxpy(volatile int *arr, int n)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) arr[i] = arr[i] + 70*i;
}

void nmc(volatile int *arr, long n)
{
	int i = 0;
	srand(time(NULL));
   for(i = 0; i <n; i+=64)
   	arr[i] = arr[i] + (int)(70*rand()/i)%100;
}

int main(){
	int max_node;
	int total_numa_nodes = 0;
	int * numa_node_ids;
	max_node = numa_max_node() + 1;
	int cpu_count = numa_num_possible_cpus();
	numa_node_ids = (int*)malloc(sizeof(int)*max_node);
	struct bitmask * numa_nodes = numa_get_membind();
	size_t * fr;
	size_t * tot;
	int n = 0;
	while(n < numa_nodes->size){
		if(numa_bitmask_isbitset(numa_nodes, n)){
			numa_node_ids[total_numa_nodes] = n;
			total_numa_nodes++;
		}
		n++;
	}

	hipDeviceProp_t cdp;
	int mt;

        volatile int * arr1;
        volatile int * arr2;
	int j=-1;
	int k= 0;
	int err=-1;
	long numa_avail;
	size_t total = 0;
	n = 0;
	hipGetDeviceCount(&j);
	fr = (size_t *)malloc(j*sizeof(size_t));
	tot = (size_t *)malloc(j*sizeof(size_t));
	for(k= 0; k < j; k++){
		hipSetDevice(k);
		hipMemGetInfo(&fr[k], &tot[k]);
		if(tot[k] > total)
			total = tot[k];
	}
	while(n < total_numa_nodes){
		numa_avail = numa_node_size(numa_node_ids[n], NULL);
		if(numa_avail < total){
			arr1 = (volatile int *)numa_alloc_onnode(numa_avail/2, numa_node_ids[n]);
			nmc(arr1, numa_avail/(2*sizeof(int)));
		}else{
			arr1 = (volatile int *)numa_alloc_onnode(total/2, numa_node_ids[n]);
			nmc(arr1, total/(2*sizeof(int)));
		}
		for(k=0;k<j;k++){
			hipSetDevice(k);
			hipGetDeviceProperties(&cdp, k);
			mt = cdp.maxThreadsPerBlock;
			err = hipMalloc(&arr2, (tot[k]/2));
			if((err == 2)){
				printf("GPGPU: %d, Numa id: %d\n",k, numa_node_ids[n] );
			}
			saxpy<<<mt/32, mt>>>(arr2, (tot[k]/2)/sizeof(int));
       			hipFree((void *)arr2);
		}

		if(numa_avail < total)
			numa_free((void *)arr1, numa_avail/2);
		else
			numa_free((void *)arr1, total/2);
		n++;
	}
        return 1;
}
