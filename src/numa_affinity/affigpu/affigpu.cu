#include "hip/hip_runtime.h"
#include "affigpu.h"

struct numa_node_bw{
	int numa_id;
	int device_id;
	char * mem_type;
	long double wr_only_avg;
        long double owtr_avg;
	struct numa_node_bw * next;
};

struct numa_node_bw * numa_node_list=NULL;
struct numa_node_bw * numa_list_head=NULL;
int mem_types;
int max_node;
int numt;
int total_numa_nodes=0;
int * numa_node_ids;
struct bitmask * numa_nodes;
char ** mem_tech;
long double * means;
int * cluster_sizes;
char classes[3][8] = {"fast", "slow", "slowest"};

__global__ void gpu_buffer(double *a, double *b, double *c, int n){
	int j;
	for(j =0; j < n; j++){
		a[j] = c[j] + b[j];
	}
}


void label_mem(){
	struct numa_node_bw * bw_it = numa_list_head;
	struct numa_node_bw * next_bw_it = bw_it->next;
	int i = 0;
	bw_it->mem_type = classes[i];
	
	while(next_bw_it != NULL){
		long double diff = bw_it->owtr_avg - next_bw_it->owtr_avg;
		long double perct = 0.2*bw_it->owtr_avg;
		if((diff > perct)&&((i+1)<3)){
			i++;
		}
		next_bw_it->mem_type = classes[i];
		bw_it = next_bw_it;
		next_bw_it= bw_it->next;
	}
}

void sort_list(struct numa_node_bw * new_node){
	struct numa_node_bw * bw_it = numa_list_head;
	struct numa_node_bw * prev_bw_it = NULL;
	while(bw_it != NULL){
		if((bw_it->owtr_avg < new_node->owtr_avg)){
			if(prev_bw_it == NULL){
				new_node->next = bw_it;
				numa_list_head = new_node;
			}else{
				prev_bw_it->next = new_node;
				new_node->next = bw_it;
			}
			return;
		}
		prev_bw_it = bw_it;
		bw_it = bw_it->next;
	}
	prev_bw_it->next = new_node;
	return;

}

void write_config_file(){
	FILE * conf;
	char fname[50];
	strcpy(fname, "numa_class");
	conf = fopen(fname, "w");
	struct numa_node_bw * bw_it = numa_list_head;
	printf("GPU ID\tNUMA ID\tType\tInit(Mb/s)\tTriad(Mb/s)\n");
	while(bw_it != NULL){	
		fprintf(conf, "%d %d %s %Lf %Lf\n", bw_it->device_id, bw_it->numa_id, bw_it->mem_type, bw_it->wr_only_avg, bw_it->owtr_avg);
		printf("%d\t%d\t%s\t%Lf\t%Lf\n",bw_it->device_id, bw_it->numa_id, bw_it->mem_type, bw_it->wr_only_avg, bw_it->owtr_avg);
		bw_it = bw_it->next;
	}
	fclose(conf);
}

void gputest(int argc, char ** argv){
	max_node = numa_max_node() + 1;
	int cpu_count = numa_num_possible_cpus();
	numa_node_ids = (int*)malloc(sizeof(int)*max_node);
	struct bitmask * numa_nodes = numa_get_membind();
	int i = 0;
	while(i < numa_nodes->size){
		if(numa_bitmask_isbitset(numa_nodes, i)){
			numa_node_ids[total_numa_nodes] = i;
			total_numa_nodes++;
		}
		i++;
	}

	int ndevs;
	hipGetDeviceCount(&ndevs);
	int mbs = 64;
	unsigned long size = mbs*1024*1024;
	double *a, *b, *c, *ag, *bg, *cg;
	struct timespec begin, stop;
	srand(clock());
	//sleep(10);
 
 	i = 0;
	while(i < ndevs){
		hipSetDevice(i);
		int iters = 0;
		int k=0;
		long double wr_only_avg=0.0;
		long double owtr_avg=0.0;
		long double accum;
	while(k < total_numa_nodes){
		for( iters = 0; iters < 10; iters++){
			int j = 0;
			a = (double*)numa_alloc_onnode(size, numa_node_ids[k]);
			b = (double*)numa_alloc_onnode(size, numa_node_ids[k]);
			c = (double*)numa_alloc_onnode(size, numa_node_ids[k]);
			hipMalloc(&ag, size);
			hipMalloc(&bg, size);
			hipMalloc(&cg, size);
			long double empty=0.0;
			clock_gettime( CLOCK_MONOTONIC, &begin);
			for(j = 0;j < (size/sizeof(double));j++){
				a[j] = 1.0;
				b[j] = 2.0;
				c[j] = 3.0;
			}
			hipMemcpy(ag, a, size, hipMemcpyHostToDevice);
			hipMemcpy(bg, b, size, hipMemcpyHostToDevice);
			hipMemcpy(cg, c, size, hipMemcpyHostToDevice);
			clock_gettime( CLOCK_MONOTONIC, &stop);
			accum = ( stop.tv_sec - begin.tv_sec ) + (long double)( stop.tv_nsec - begin.tv_nsec ) / (long double)BILLION;
			wr_only_avg += ((3*size*1.0E-06)/(long double)(accum - empty));
			clock_gettime( CLOCK_MONOTONIC, &begin);
			gpu_buffer<<<((size/sizeof(double))+255)/256, 256>>>(ag,bg,cg,(size/sizeof(double)));
			clock_gettime( CLOCK_MONOTONIC, &stop);
			accum = ( stop.tv_sec - begin.tv_sec ) + (long double)( stop.tv_nsec - begin.tv_nsec ) / (long double)BILLION;
			owtr_avg += ((3*size*1.0E-06)/(long double)(accum - empty));
			hipFree(ag);
			hipFree(bg);
			hipFree(cg);
			numa_free(a, size);
			numa_free(b, size);
			numa_free(c, size);
		}
		struct numa_node_bw * node_bw = (struct numa_node_bw *)malloc(sizeof(struct numa_node_bw));
		node_bw->device_id = i;
		node_bw->numa_id = numa_node_ids[k];
		node_bw->wr_only_avg = wr_only_avg/10;
		node_bw->owtr_avg = owtr_avg/10;
		node_bw->next = NULL;
		if(numa_node_list == NULL){
			numa_node_list = node_bw;
			numa_list_head = numa_node_list;
		}
		else{
			sort_list(node_bw);
		}
		k++;
	}
		i++;
	}
	label_mem();
	write_config_file();
}
